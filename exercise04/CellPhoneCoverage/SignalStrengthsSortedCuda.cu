#include "hip/hip_runtime.h"
#include "SignalStrengthsSortedCuda.h"

#include "CellPhoneCoverage.h"
#include "CudaArray.h"
#include "Helpers.h"

#include <iostream>
using namespace std;

// "Smart" CUDA implementation which computes signal strengths
//
// First, all transmitters are sorted into buckets
// Then, all receivers are sorted into buckets
// Then, receivers only compute signal strength against transmitters in nearby buckets
//
// This multi-step algorithm makes the signal strength computation scale much
//  better to high number of transmitters/receivers

struct Bucket
{
	int startIndex; // Start of bucket within array
	int numElements; // Number of elements in bucket
};

///////////////////////////////////////////////////////////////////////////////////////////////
//
// No-operation sorting kernel
//
// This takes in an unordered set, and builds a dummy bucket representation around it
// It does not perform any actual sorting!
//
// This kernel must be launched with a 1,1 configuration (1 grid block, 1 thread).

static __global__ void noSortKernel(const Position* inputPositions,
		int numInputPositions, Position* outputPositions, Bucket* outputBuckets)
{
	int numBuckets = BucketsPerAxis * BucketsPerAxis;

	// Copy contents of input positions into output positions

	for (int i = 0; i < numInputPositions; ++i)
		outputPositions[i] = inputPositions[i];

	// Set up the set of buckets to cover the output positions evenly

	for (int i = 0; i < numBuckets; i++)
	{
		Bucket& bucket = outputBuckets[i];

		bucket.startIndex = numInputPositions * i / numBuckets;
		bucket.numElements = (numInputPositions * (i + 1) / numBuckets)
				- bucket.startIndex;
	}
}

// !!! missing !!!
// Kernels needed for sortPositionsIntoBuckets(...)

///////////////////////////////////////////////////////////////////////////////////////////////
//
// Sort a set of positions into a set of buckets
//
// Given a set of input positions, these will be re-ordered such that
//  each range of elements in the output array belong to the same bucket.
// The list of buckets that is output describes where each such range begins
//  and ends in the re-ordered position array.

static void sortPositionsIntoBuckets(CudaArray<Position>& cudaInputPositions,
		CudaArray<Position>& cudaOutputPositions,
		CudaArray<Bucket>& cudaOutputPositionBuckets)
{
	// Bucket sorting with "Counting Sort" is a multi-phase process:
	//
	// 1. Determine how many of the input elements should end up in each bucket (build a histogram)
	//
	// 2. Given the histogram, compute where in the output array that each bucket begins, and how large it is
	//    (perform prefix summation over the histogram)
	//
	// 3. Given the start of each bucket within the output array, scatter elements from the input
	//    array into the output array
	//
	// Your new sort implementation should be able to handle at least 10 million entries, and
	//  run in reasonable time (the reference implementations does the job in less than 5 seconds).

	//=================  Your code here =====================================
	// !!! missing !!!

	// Instead of sorting, we will now run a dummy kernel that just duplicates the
	//  output positions, and constructs a set of dummy buckets. This is just so that
	//  the test program will not crash when you try to run it.
	//
	// This kernel is run single-threaded because it is throw-away code where performance
	//  does not matter; after all, the purpose of the lab is to replace it with a
	//  proper sort algorithm instead!

	//=========== Remove this code when you begin to implement your own sorting algorithm =================

	noSortKernel<<<1, 1>>>(cudaInputPositions.hipArray(),
			cudaInputPositions.size(), cudaOutputPositions.hipArray(),
			cudaOutputPositionBuckets.hipArray());

}

///////////////////////////////////////////////////////////////////////////////////////////////
//
// Go through all transmitters in one bucket, find highest signal strength
// Return highest strength (or the old value, if that was higher)

static __device__ float scanBucket(const Position* transmitters,
		int numTransmitters, const Position& receiver, float bestSignalStrength)
{
	for (int transmitterIndex = 0; transmitterIndex < numTransmitters;
			++transmitterIndex)
	{
		const Position& transmitter = transmitters[transmitterIndex];

		float strength = signalStrength(transmitter, receiver);

		if (bestSignalStrength < strength)
			bestSignalStrength = strength;
	}

	return bestSignalStrength;
}

///////////////////////////////////////////////////////////////////////////////////////////////
//
// Calculate signal strength for all receivers

static __global__ void calculateSignalStrengthsSortedKernel(
		const Position* transmitters, const Bucket* transmitterBuckets,
		const Position* receivers, const Bucket* receiverBuckets,
		float* signalStrengths)
{
	// Determine which bucket the current grid block is processing

	int receiverBucketIndexX = blockIdx.x;
	int receiverBucketIndexY = blockIdx.y;

	int receiverBucketIndex = receiverBucketIndexY * BucketsPerAxis
			+ receiverBucketIndexX;

	const Bucket& receiverBucket = receiverBuckets[receiverBucketIndex];

	int receiverStartIndex = receiverBucket.startIndex;
	int numReceivers = receiverBucket.numElements;

	// Distribute available receivers over the set of available threads

	for (int receiverIndex = threadIdx.x; receiverIndex < numReceivers;
			receiverIndex += blockDim.x)
	{
		// Locate current receiver within the current bucket

		const Position& receiver = receivers[receiverStartIndex + receiverIndex];
		float& finalStrength = signalStrengths[receiverStartIndex
				+ receiverIndex];

		float bestSignalStrength = 0.f;

		// Scan all buckets in the 3x3 region enclosing the receiver's bucket index

		for (int transmitterBucketIndexY = receiverBucketIndexY - 1;
				transmitterBucketIndexY < receiverBucketIndexY + 2;
				++transmitterBucketIndexY)
			for (int transmitterBucketIndexX = receiverBucketIndexX - 1;
					transmitterBucketIndexX < receiverBucketIndexX + 2;
					++transmitterBucketIndexX)
			{
				// Only process bucket if its index is within [0, BucketsPerAxis - 1] along each axis

				if (transmitterBucketIndexX >= 0
						&& transmitterBucketIndexX < BucketsPerAxis
						&& transmitterBucketIndexY >= 0
						&& transmitterBucketIndexY < BucketsPerAxis)
				{
					// Scan bucket for a potential new "highest signal strength"

					int transmitterBucketIndex = transmitterBucketIndexY
							* BucketsPerAxis + transmitterBucketIndexX;
					int transmitterStartIndex =
							transmitterBuckets[transmitterBucketIndex].startIndex;
					int numTransmitters =
							transmitterBuckets[transmitterBucketIndex].numElements;
					bestSignalStrength = scanBucket(
							&transmitters[transmitterStartIndex],
							numTransmitters, receiver, bestSignalStrength);
				}
			}

		// Store out the highest signal strength found for the receiver

		finalStrength = bestSignalStrength;
	}
}

///////////////////////////////////////////////////////////////////////////////////////////////

void calculateSignalStrengthsSortedCuda(const PositionList& cpuTransmitters,
		const PositionList& cpuReceivers,
		SignalStrengthList& cpuSignalStrengths)
{
	int numBuckets = BucketsPerAxis * BucketsPerAxis;

	// Copy input positions to device memory

	CudaArray<Position> cudaTempTransmitters(cpuTransmitters.size());
	cudaTempTransmitters.copyToCuda(&(*cpuTransmitters.begin()));

	CudaArray<Position> cudaTempReceivers(cpuReceivers.size());
	cudaTempReceivers.copyToCuda(&(*cpuReceivers.begin()));

	// Allocate device memory for sorted arrays

	CudaArray<Position> cudaTransmitters(cpuTransmitters.size());
	CudaArray<Bucket> cudaTransmitterBuckets(numBuckets);

	CudaArray<Position> cudaReceivers(cpuReceivers.size());
	CudaArray<Bucket> cudaReceiverBuckets(numBuckets);

	// Sort transmitters and receivers into buckets

	sortPositionsIntoBuckets(cudaTempTransmitters, cudaTransmitters,
			cudaTransmitterBuckets);
	sortPositionsIntoBuckets(cudaTempReceivers, cudaReceivers,
			cudaReceiverBuckets);

	// Perform signal strength computation
	CudaArray<float> cudaSignalStrengths(cpuReceivers.size());

	int numThreads = 256;
	dim3 grid = dim3(BucketsPerAxis, BucketsPerAxis);

	calculateSignalStrengthsSortedKernel<<<grid, numThreads>>>(
			cudaTransmitters.hipArray(), cudaTransmitterBuckets.hipArray(),
			cudaReceivers.hipArray(), cudaReceiverBuckets.hipArray(),
			cudaSignalStrengths.hipArray());

	// Copy results back to host memory
	cpuSignalStrengths.resize(cudaSignalStrengths.size());
	cudaSignalStrengths.copyFromCuda(&(*cpuSignalStrengths.begin()));
}
