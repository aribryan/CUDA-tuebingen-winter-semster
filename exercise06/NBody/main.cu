#include "hip/hip_runtime.h"
#include "gltools.h"
#include "Tools.h"

#include <iostream>
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>

using namespace std;

#define GUI
#define NUM_FRAMES 250

#define THREADS_PER_BLOCK 128
#define EPS_2 0.00001f
#define GRAVITY 0.00000001f

float randF(const float min = 0.0f, const float max = 1.0f)
{
	int randI = rand();
	float randF = (float) randI / (float) RAND_MAX;
	float result = min + randF * (max - min);

	return result;
}

inline __device__ float2 operator+(const float2 op1, const float2 op2)
{
	return make_float2(op1.x + op2.x, op1.y + op2.y);
}

inline __device__ float2 operator-(const float2 op1, const float2 op2)
{
	return make_float2(op1.x - op2.x, op1.y - op2.y);
}

inline __device__ float2 operator*(const float2 op1, const float op2)
{
	return make_float2(op1.x * op2, op1.y * op2);
}

inline __device__ float2 operator/(const float2 op1, const float op2)
{
	return make_float2(op1.x / op2, op1.y / op2);
}

inline __device__ void operator+=(float2 &a, const float2 b)
{
	a.x += b.x;
	a.y += b.y;
}


__global__ void acceleration(float2 *acc, float2 *pos, float *mass, int n)
{
	//extern __shared__ float temp[];

	int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x<n)
	{
		float2 myPos = pos[x];
		float2 sum;
		sum.x = 0;
		sum.y = 0;
		for (int i=0;i<n;i++) {
			float2 position = pos[i];

			float2 dis = (position-myPos);
			float l2_norm = sqrtf(powf((position.x-myPos.x),2) + powf((position.y-myPos.y),2));
			sum += (dis * mass[i])/powf(((l2_norm * l2_norm) + EPS_2),1.5);
			//sum.y += (mass[i] * dis.y)/powf(((l2_norm * l2_norm) + EPS_2),1.5);
		}
		sum = sum * GRAVITY;
		acc[x].x = sum.x;
		acc[x].y = sum.y;
	}
}

__global__ void velocity(float2 *vel, float2 *acc,int n)
{
	//extern __shared__ float temp[];

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x<n){
		vel[x]+=acc[x];}
}

__global__ void positionKern(float2 *pos, float2 *vel,int n)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x<n){
		pos[x]+=vel[x];}
}



int main(int argc, char **argv)
{
	if (argc != 2)
	{
		cout << "Usage: " << argv[0] << " <numBodies>" << endl;
		return 1;
	}
	unsigned int numBodies = atoi(argv[1]);
	unsigned int numBlocks = numBodies / THREADS_PER_BLOCK;
	numBodies = numBlocks * THREADS_PER_BLOCK;

	// allocate memory
	float2* hPositions = new float2[numBodies];
	float2* hVelocities = new float2[numBodies];
	float* hMasses = new float[numBodies];

	// Initialize Positions and speed
	for (unsigned int i = 0; i < numBodies; i++)
	{
		hPositions[i].x = randF(-1.0, 1.0);
		hPositions[i].y = randF(-1.0, 1.0);
		hVelocities[i].x = hPositions[i].y * 0.007f + randF(0.001f, -0.001f);
		hVelocities[i].y = -hPositions[i].x * 0.007f + randF(0.001f, -0.001f);
		hMasses[i] = randF(0.0f, 1.0f) * 10000.0f / (float) numBodies;
	}

	// float
	float2* gPositions;
	float2* gVelocities;
	float* gMasses;
	float2* gAcc;
	hipMalloc((void**) &gPositions, (numBodies) * sizeof(float2));
	hipMalloc((void**) &gVelocities, (numBodies) * sizeof(float2));
	hipMalloc((void**) &gMasses, (numBodies) * sizeof(float));
	hipMalloc((void**) &gAcc, (numBodies) * sizeof(float2));

	hipMemcpy(gPositions,hPositions,numBodies*sizeof(float2),hipMemcpyHostToDevice);
	hipMemcpy(gVelocities,hVelocities,numBodies*sizeof(float2),hipMemcpyHostToDevice);
	hipMemcpy(gMasses,hMasses,numBodies*sizeof(float),hipMemcpyHostToDevice);


	//hipArray* gPositions;
	//hipChannelFormatDesc floatTex1 = hipCreateChannelDesc<float2>();
	//hipMallocArray(&gPositions, &floatTex1, numBodies);

	//hipArray* gMasses;
	//hipChannelFormatDesc floatTex2 = hipCreateChannelDesc<float>();
	//hipMallocArray(&gMasses, &floatTex2, numBodies);

	//hipMemcpyToArray(gPositions, 0, 0, hPositions, numBodies * sizeof(float2),
	//			hipMemcpyHostToDevice);
	//hipMemcpyToArray(gMasses, 0, 0, hMasses, numBodies * sizeof(float),
	//				hipMemcpyHostToDevice);



	// TODO 1: Allocate GPU memory for
	// - Positions,
	// - Velocities,
	// - Accelerations and
	// - Masses
	// of all bodies and initialize them from the CPU arrays (where available).

	// Free host memory not needed again

	delete[] hMasses;

	// Initialize OpenGL rendering
#ifdef GUI
	initGL();
	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
	GLuint sp = createShaderProgram("white.vs", 0, 0, 0, "white.fs");

	GLuint vb;
	glGenBuffers(1, &vb);
	GL_CHECK_ERROR;
	glBindBuffer(GL_ARRAY_BUFFER, vb);
	GL_CHECK_ERROR;
	glBufferData(GL_ARRAY_BUFFER, sizeof(float) * 2 * numBodies, hPositions,
			GL_STATIC_DRAW);
	GL_CHECK_ERROR;
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	GL_CHECK_ERROR;

	GLuint va;
	glGenVertexArrays(1, &va);
	GL_CHECK_ERROR;
	glBindVertexArray(va);
	GL_CHECK_ERROR;
	glBindBuffer(GL_ARRAY_BUFFER, vb);
	GL_CHECK_ERROR;
	glEnableVertexAttribArray(glGetAttribLocation(sp, "inPosition"));
	GL_CHECK_ERROR;
	glVertexAttribPointer(glGetAttribLocation(sp, "inPosition"), 2, GL_FLOAT,
			GL_FALSE, 0, 0);
	GL_CHECK_ERROR;
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	GL_CHECK_ERROR;
	glBindVertexArray(0);
	GL_CHECK_ERROR;
#endif

	// Calculate
	for(unsigned int t = 0; t < NUM_FRAMES; t++)
	{
		__int64_t computeStart = continuousTimeNs();

		// TODO 3: Update accelerations of all bodies here.
		acceleration<<<numBlocks, THREADS_PER_BLOCK>>>(gAcc, gPositions,gMasses,numBodies);
		hipDeviceSynchronize();
		//hipMemcpy(hVelocity,gVelocity,numBodies*sizeof(float2),cudaDeviceToHost);

		velocity<<<numBlocks, THREADS_PER_BLOCK>>>(gVelocities, gAcc,numBodies);
		hipDeviceSynchronize();
		positionKern<<<numBlocks, THREADS_PER_BLOCK>>>(gPositions, gVelocities,numBodies);
		hipDeviceSynchronize();
		hipMemcpy(hPositions,gPositions,numBodies*sizeof(float2),hipMemcpyDeviceToHost);

		// TODO 4: Update velocities and positions of all bodies here.


		hipDeviceSynchronize();
		cout << "Frame compute time: " << (continuousTimeNs() - computeStart)
				<< "ns" << endl;

		// TODO 5: Download the updated positions into the hPositions array for rendering.

#ifdef GUI
		// Upload positions to OpenGL
		glBindBuffer(GL_ARRAY_BUFFER, vb);
		GL_CHECK_ERROR;
		glBufferData(GL_ARRAY_BUFFER, sizeof(float) * 2 * numBodies, hPositions,
				GL_STATIC_DRAW);
		GL_CHECK_ERROR;
		glBindBuffer(GL_ARRAY_BUFFER, 0);
		GL_CHECK_ERROR;

		// Draw
		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
		GL_CHECK_ERROR;
		glUseProgram(sp);
		GL_CHECK_ERROR;
		glBindVertexArray(va);
		GL_CHECK_ERROR;
		glDrawArrays(GL_POINTS, 0, numBodies);
		GL_CHECK_ERROR;
		glBindVertexArray(0);
		GL_CHECK_ERROR;
		glUseProgram(0);
		GL_CHECK_ERROR;
		swapBuffers();
#endif
	}

#ifdef GUI
	cout << "Done." << endl;
	sleep(2);
#endif

	// Clean up
#ifdef GUI
	glDeleteProgram(sp);
	GL_CHECK_ERROR;
	glDeleteVertexArrays(1, &va);
	GL_CHECK_ERROR;
	glDeleteBuffers(1, &vb);
	GL_CHECK_ERROR;

	glDeleteProgram(sp);
	exitGL();
#endif

	// TODO 2: Clean up your allocated memory

	delete[] hPositions;
	delete[] hVelocities;
	hipFree(gAcc);
	hipFree(gPositions);
	hipFree(gVelocities);
	hipFree(gMasses);

}

